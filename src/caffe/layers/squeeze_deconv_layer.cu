#include "hip/hip_runtime.h"
/***************************** MulticoreWare_Modified - Feature: Pruning / Splicing ************************************/
#include <vector>
#include <cmath>
#include<stdio.h>

#include "caffe/filler.hpp"
#include "caffe/layers/squeeze_deconv_layer.hpp"

namespace caffe {

// The constant NUM_THREADS should be equal to the value in SqueezeCMomentCalc
template <typename Dtype>
__global__ void SqueezeCMomentCollect(const int n, const Dtype* wb, const Dtype* mask,
    Dtype* mu, Dtype* std, unsigned int* count ) {
  const int NUM_THREADS = 512;
  __shared__ Dtype param [4 * NUM_THREADS];
  __shared__ unsigned int tcount [2 * NUM_THREADS];
  unsigned int t = threadIdx.x;
  unsigned int s = 2 * blockIdx.x * NUM_THREADS;
  if (s + t < n){
    param[t] = fabs(mask[s + t] * wb[s + t]);
    param[t + 2 * NUM_THREADS] = mask[s + t] * wb[s + t] * wb[s + t];
    if(mask[s + t] * wb[s + t] != 0) tcount[t] = 1;
    else tcount[t] = 0;
  }
  else{
    param[t] = 0;param[t + 2 * NUM_THREADS] = 0;tcount[t] = 0;
  }
  if (s + t + NUM_THREADS < n){
    param[t + NUM_THREADS] = fabs(mask[s + t + NUM_THREADS] * wb[s + t + NUM_THREADS]);
    param[t + 3 * NUM_THREADS] = mask[s + t + NUM_THREADS] * wb[s + t + NUM_THREADS] * wb[s + t + NUM_THREADS];
    if(mask[s + t +NUM_THREADS] * wb[s + t + NUM_THREADS] != 0) tcount[t + NUM_THREADS] = 1;
    else tcount[ t + NUM_THREADS] = 0;
  }
  else{
    param[t + NUM_THREADS] = 0; param[t + 3 * NUM_THREADS] = 0; tcount[t + NUM_THREADS] = 0;
  }
  __syncthreads();
  for(unsigned int stride = NUM_THREADS; stride >= 1; stride >>= 1) {
    if (t < stride ){
      param[t] += param[t + stride];
      param[t + 2 * NUM_THREADS] += param[t + 2 * NUM_THREADS + stride];
      tcount[t] += tcount[t + stride];
    }
    __syncthreads();
  }
  if (t == 0){
    mu   [blockIdx.x] = param[0];
    std  [blockIdx.x] = param[2 * NUM_THREADS];
    count[blockIdx.x] = tcount[0];
  }
}

// The constant NUM_THREADS should be equal to the value in SqueezeCMomentCalc
template <typename Dtype>
__global__ void SqueezeCNzeroCollect(const int n, const Dtype* mask, unsigned int* count ) {
  const int NUM_THREADS = 512;
  __shared__ unsigned int tcount [2 * NUM_THREADS];
  unsigned int t = threadIdx.x;
  unsigned int s = 2 * blockIdx.x * NUM_THREADS;
  tcount[t] = 0;
  if (s + t < n && mask[s + t] != 0){
    tcount[t] = 1;
  }
  tcount[t+NUM_THREADS] = 0;
  if (s + t + NUM_THREADS < n && mask[s + t + NUM_THREADS] != 0){
    tcount[t + NUM_THREADS] = 1;
  }
  __syncthreads();
  for(unsigned int stride = NUM_THREADS; stride >= 1; stride >>= 1) {
    if (t < stride ){
      tcount[t] += tcount[t + stride];
    }
    __syncthreads();
  }
  if (t == 0){
    count[blockIdx.x] = tcount[0];
  }
}

//Check condition for pruning and splicing
template <typename Dtype>
__global__ void SqueezeCMaskCalc(const int n, const Dtype* wb,
    Dtype* mask, Dtype mu, Dtype std, Dtype r) {
  CUDA_KERNEL_LOOP(index, n) {
    // The constants 0.9 and 1.1 is to set margin that witholds few parameters undergoing pruning / splicing
    if (mask[index] > 0 && fabs(wb[index]) <= 0.9 * r * max(mu + std, Dtype(0))) {
      mask[index] = 0;
    }
    else if (mask[index] == 0 && fabs(wb[index]) > 1.1 * r * max(mu + std, Dtype(0)) && r !=0 ){
      mask[index] = 1;
    }
  }
}

template <typename Dtype>
__global__ void SqueezeCMaskApply(const int n, const Dtype* wb,
    const Dtype* mask, Dtype* wb_t) {
  CUDA_KERNEL_LOOP(index, n) {
    wb_t[index] = wb[index] * mask[index];
  }
}

template <typename Dtype>
__global__ void ValidateMask(const int n,  Dtype* wb) {
  CUDA_KERNEL_LOOP(index, n) {
  if (wb[index] !=0 && wb[index]!= 1)
    wb[index] = fabs(rintf(wb[index]));
  }
}

//Calculate Mean and std deviation of weights
template <typename Dtype>
void SqueezeCMomentCalc(const int n, const Dtype* wb, const Dtype* mask, Dtype* mu, Dtype* std, unsigned int* ncount){
  const unsigned int NUM_THREADS = 512;
  Dtype* pmu_g; Dtype* pstd_g; unsigned int* pncount_g;
  Dtype* pmu_c; Dtype* pstd_c; unsigned int* pncount_c;
  int num_p = (n + (NUM_THREADS << 1) - 1) / (NUM_THREADS << 1);
  hipMalloc(&pmu_g, sizeof(Dtype)  * num_p);
  hipMalloc(&pstd_g, sizeof(Dtype) * num_p);
  hipMalloc(&pncount_g, sizeof(unsigned int) * num_p);
  pmu_c = (Dtype*) malloc(num_p * sizeof(Dtype));
  pstd_c = (Dtype*) malloc(num_p * sizeof(Dtype));
  pncount_c = (unsigned int*) malloc(num_p * sizeof(unsigned int));
  SqueezeCMomentCollect<Dtype><<<num_p,NUM_THREADS>>>(n, wb, mask, pmu_g, pstd_g, pncount_g);
  CUDA_POST_KERNEL_CHECK;
  hipMemcpy(pmu_c, pmu_g, sizeof(Dtype) * num_p, hipMemcpyDeviceToHost);
  hipMemcpy(pstd_c, pstd_g, sizeof(Dtype) * num_p, hipMemcpyDeviceToHost);
  hipMemcpy(pncount_c, pncount_g, sizeof(unsigned int) * num_p, hipMemcpyDeviceToHost);
  for (int i = 0; i < num_p; i++) {
    *mu += pmu_c[i]; *std += pstd_c[i]; *ncount += pncount_c[i];
  }
  hipFree(pmu_g);hipFree(pstd_g);hipFree(pncount_g);
  free(pmu_c);free(pstd_c);free(pncount_c);
}

template <typename Dtype>
void SqueezeDeconvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* weight = NULL;
  Dtype* weightMask = NULL;
  Dtype* weightTmp = NULL;
  const Dtype* bias = NULL;
  Dtype* biasMask = NULL;
  Dtype* biasTmp = NULL;
  Dtype* prune_threshold_params_gpu = NULL; // To store mu and std values
  Dtype* prune_threshold_params_cpu = NULL;
  prune_threshold_params_cpu = (Dtype*)malloc(sizeof(Dtype) * 2);
  int maskcount = 0;
  if (this->bias_term_) {
    weight = this->blobs_[0]->mutable_gpu_data();
    weightMask = this->blobs_[2]->mutable_gpu_data();
    weightTmp = this->weight_tmp_.mutable_gpu_data();
    bias = this->blobs_[1]->mutable_gpu_data();
    biasMask = this->blobs_[3]->mutable_gpu_data();
    prune_threshold_params_gpu = this->blobs_[4]->mutable_gpu_data();
    biasTmp = this->bias_tmp_.mutable_gpu_data();
    maskcount = this->blobs_[2]->count();
  }
  else {
    weight = this->blobs_[0]->mutable_gpu_data();
    weightMask = this->blobs_[1]->mutable_gpu_data();
    prune_threshold_params_gpu = this->blobs_[2]->mutable_gpu_data();
    weightTmp = this->weight_tmp_.mutable_gpu_data();
    maskcount = this->blobs_[1]->count();
  }

  if (this->phase_ == TRAIN) {

      // Validate mask value to avoid corrupted mask value
    ValidateMask<Dtype><<<CAFFE_GET_BLOCKS(maskcount),
    CAFFE_CUDA_NUM_THREADS>>>(maskcount,weightMask);
    CUDA_POST_KERNEL_CHECK;

    // Calculate the mean and standard deviation of learnable parameters
    if ((this->std == 0 && this->iter_ == 0) || this->iter_== 40 || this->iter_== 80 || this->iter_== 120 || this->iter_== 160) {
      unsigned int ncount = 0;
      SqueezeCMomentCalc(this->blobs_[0]->count(), weight, weightMask, &this->mu, &this->std, &ncount);
      if (this->bias_term_) {
        SqueezeCMomentCalc(this->blobs_[1]->count(), bias, biasMask, &this->mu, &this->std, &ncount);
      }
      this->mu /= ncount; this->std -= ncount * this->mu * this->mu;
      this->std /= ncount; this->std = sqrt(this->std);
      prune_threshold_params_cpu[0] = this->mu;
      prune_threshold_params_cpu[1] = this->std;
      LOG(INFO)<<mu<<"  "<<std<<"  "<<ncount<<"\n";
      // Copy mu and std value from host to device
      hipMemcpy(prune_threshold_params_gpu, prune_threshold_params_cpu, sizeof(Dtype)*2, hipMemcpyHostToDevice);
    }
    // Copy mu and std value from Device to host
    hipMemcpy(prune_threshold_params_cpu, prune_threshold_params_gpu, sizeof(Dtype)*2, hipMemcpyDeviceToHost);
    //No pruning/splicing during Retraining
    // Calculate the weight mask and bias mask with probability
    Dtype r = static_cast<Dtype>(rand())/static_cast<Dtype>(RAND_MAX);
    if (pow(1 + (this->gamma) * (this->iter_), -(this->power)) > r && (this->iter_) < (this->iter_stop_)) {
      SqueezeCMaskCalc<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
        CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[0]->count(), weight,
        weightMask, prune_threshold_params_cpu[0], prune_threshold_params_cpu[1], this->crate);

      CUDA_POST_KERNEL_CHECK;
      if (this->bias_term_) {
        SqueezeCMaskCalc<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[1]->count()),
          CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[1]->count(), bias,
          biasMask, prune_threshold_params_cpu[0], prune_threshold_params_cpu[1], this->crate);
        CUDA_POST_KERNEL_CHECK;
      }
    }
//Dynamic Splicing
//Randomly unprune the pruned weights based on the splicing ratio
  if(this->dynamicsplicing)
  {
    if (this->iter_ == 0) {
        Dtype* weight_cpu = (Dtype *)malloc(this->blobs_[0]->count() *(sizeof(Dtype)));
        Dtype* weightMask_cpu = (Dtype *)malloc(this->blobs_[0]->count() *(sizeof(Dtype)));
        // Initially copy weight, weightMask to weight_cpu, weightMask_cpu and do Dynamic Splicing
        hipMemcpy(weight_cpu, weight, this->blobs_[0]->count() *(sizeof(Dtype)), hipMemcpyDeviceToHost);
        hipMemcpy(weightMask_cpu, weightMask, this->blobs_[0]->count() *(sizeof(Dtype)), hipMemcpyDeviceToHost);
        // Vector Pair holds weights and corresponding index for pruned nodes
        std::vector<std::pair<float, int> > prune_node;
        for (unsigned int k = 0; k < this->blobs_[0]->count(); ++k) {
          if(weightMask_cpu[k] == 0) {
            prune_node.push_back(make_pair(fabs(weight_cpu[k]), k));
          }
        }
        // Sort the weights and unprune the nodes
        std::sort(prune_node.begin(), prune_node.end());
        int zero_count = prune_node.size();
        int to_bespliced = zero_count * this->splicing_rate;
        int start_index = 0;
        int end_index = 0;
        for (unsigned int k = 0; k < zero_count; ++k) {
          if (prune_node[k].first > (0.25 * (prune_threshold_params_cpu[0] + prune_threshold_params_cpu[1]))) {
            start_index = k;
            break;
          }
        }
        if(start_index == 0)
          start_index = zero_count - to_bespliced;  //Update start index
        end_index = start_index + to_bespliced;
        if (end_index > zero_count) {
          start_index = start_index - (end_index - zero_count);
          end_index = start_index + to_bespliced;
        }
        for (unsigned int k = start_index; k < end_index; ++k) {
          weightMask_cpu[prune_node[k].second] = 1;
        }
        hipMemcpy(weightMask, weightMask_cpu, this->blobs_[0]->count() *(sizeof(Dtype)), hipMemcpyHostToDevice);
        free(weightMask_cpu);
        free(weight_cpu);
        this->dynamicsplicing = false;
      }
    }
    free(prune_threshold_params_cpu);
  }

  // Calculate the current (masked) weight and bias
  SqueezeCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
    CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[0]->count(), weight, weightMask, weightTmp);
  CUDA_POST_KERNEL_CHECK;
  if (this->bias_term_) {
    SqueezeCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[1]->count()),
      CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[1]->count(), bias, biasMask, biasTmp);
    CUDA_POST_KERNEL_CHECK;
  }
  // Forward calculation with (masked) weight and bias
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    for (int n = 0; n < this->num_; ++n) {
      this->backward_gpu_gemm(bottom_data + bottom[i]->offset(n), weightTmp,
          top_data + top[i]->offset(n));
      if (this->bias_term_) {
        this->forward_gpu_bias(top_data +  n * this->top_dim_, biasTmp);
      }
    }
  }
}

template <typename Dtype>
void SqueezeDeconvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weightTmp = this->weight_tmp_.gpu_data();
  const Dtype* weightMask = NULL;
  if(this->bias_term_)
    weightMask = this->blobs_[2]->gpu_data();
  else
    weightMask = this->blobs_[1]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    //const Dtype* bottom_data = bottom_data[i]->gpu_data();
    //Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      const Dtype* biasMask = this->blobs_[3]->gpu_data();
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      SqueezeCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[3]->count()),
        CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[3]->count(), bias_diff, biasMask, bias_diff);
      CUDA_POST_KERNEL_CHECK;
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + top[i]->offset(n));
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      SqueezeCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
        CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[0]->count(), weight_diff, weightMask, weight_diff);
      CUDA_POST_KERNEL_CHECK;
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(top_diff + top[i]->offset(n),
              bottom_data + bottom[i]->offset(n), weight_diff);
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->forward_gpu_gemm(top_diff + top[i]->offset(n), weightTmp,
              bottom_diff + bottom[i]->offset(n), this->param_propagate_down_[0]);
        }
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SqueezeDeconvolutionLayer);
}
