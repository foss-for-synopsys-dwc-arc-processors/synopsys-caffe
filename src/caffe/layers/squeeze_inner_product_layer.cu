#include "hip/hip_runtime.h"
/***************************** MulticoreWare_Modified - Feature: Pruning / Splicing ************************************/
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/squeeze_inner_product_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include <cmath>

namespace caffe {

// The constant NUM_THREADS should be equal to the value in SqueezeCMomentCalc
template <typename Dtype>
__global__ void SqueezeCMomentCollect(const int n, const Dtype* wb, const Dtype* mask,
    Dtype* mu, Dtype* std, unsigned int* count ) {
  const int NUM_THREADS = 512;
  __shared__ Dtype param [4*NUM_THREADS];
  __shared__ unsigned int tcount [2*NUM_THREADS];
  unsigned int t = threadIdx.x;
  unsigned int s = 2 * blockIdx.x * NUM_THREADS;
  if (s+t < n){
    param[t] = fabs(mask[s + t] * wb[s + t]);
    param[t + 2 * NUM_THREADS] = mask[s + t] * wb[s + t] * wb[s + t];
    if(mask[s + t] * wb[s + t]!= 0) tcount[t] = 1;
    else tcount[t] = 0;
  }
  else{
    param[t] = 0;param[t +2 * NUM_THREADS] = 0;tcount[t] = 0;
  }
  if (s + t + NUM_THREADS < n){
    param[t + NUM_THREADS] = fabs(mask[s + t + NUM_THREADS] * wb[s + t + NUM_THREADS]);
    param[t + 3 * NUM_THREADS] = mask[s + t + NUM_THREADS] * wb[ s + t + NUM_THREADS] *wb[s + t + NUM_THREADS];
    if(mask[s + t + NUM_THREADS] * wb[s + t + NUM_THREADS] != 0) tcount[t + NUM_THREADS] = 1;
    else tcount[t + NUM_THREADS] = 0;
  }
  else{
    param[t + NUM_THREADS] = 0;param[t + 3 * NUM_THREADS] = 0;tcount[t + NUM_THREADS] = 0;
  }
  __syncthreads();
  for(unsigned int stride = NUM_THREADS; stride >= 1; stride >>= 1) {
    if (t < stride ){
      param[t] += param[t + stride];
      param[t + 2 * NUM_THREADS] += param[t + 2 * NUM_THREADS + stride];
      tcount[t] += tcount[t+stride];
    }
    __syncthreads();
  }
  if (t == 0){
    mu   [blockIdx.x] = param[0];
    std  [blockIdx.x] = param[2*NUM_THREADS];
    count[blockIdx.x] = tcount[0];
  }
}

// The constant NUM_THREADS should be equal to the value in SqueezeCMomentCalc
template <typename Dtype>
__global__ void SqueezeCNzeroCollect(const int n, const Dtype* mask, unsigned int* count ) {
  const int NUM_THREADS = 512;
  __shared__ unsigned int tcount [2 * NUM_THREADS];
  unsigned int t = threadIdx.x;
  unsigned int s = 2 * blockIdx.x * NUM_THREADS;
  tcount[t] = 0;
  if (s + t < n && mask[s+t]!=0){
    tcount[t] = 1;
  }
  tcount[t + NUM_THREADS] = 0;
  if (s + t + NUM_THREADS < n && mask[s + t + NUM_THREADS] != 0){
    tcount[t + NUM_THREADS] = 1;
  }
  __syncthreads();
  for(unsigned int stride = NUM_THREADS; stride >= 1; stride >>= 1) {
    if (t < stride ){
      tcount[t] += tcount[t+stride];
    }
    __syncthreads();
  }
  if (t == 0){
    count[blockIdx.x] = tcount[0];
  }
}
//Condition for pruning and splicing
template <typename Dtype>
__global__ void SqueezeCMaskCalc(const int n, const Dtype* wb,
    Dtype* mask, Dtype mu, Dtype std, Dtype r) {
  CUDA_KERNEL_LOOP(index, n) {
    // The constants 0.9 and 1.1 is to set margin that witholds few parameters undergoing pruning / splicing
    if (mask[index] > 0 && fabs(wb[index]) <= 0.9 * r * max(mu + std, Dtype(0)))
      mask[index] = 0;
    else if (mask[index] == 0 && fabs(wb[index])> 1.1 * r * max(mu + std, Dtype(0)) && r!=0)
      mask[index] = 1;
  }
}

template <typename Dtype>
__global__ void SqueezeCMaskApply(const int n, const Dtype* wb,
    const Dtype* mask, Dtype* wb_t) {
  CUDA_KERNEL_LOOP(index, n) {
    wb_t[index] = wb[index] * mask[index];
  }
}

template <typename Dtype>
__global__ void ValidateMask(const int n,  Dtype* wb) {
  CUDA_KERNEL_LOOP(index, n) {
  if (wb[index] !=0 && wb[index]!= 1)
    wb[index] = fabs(rintf(wb[index]));
  }
}

template <typename Dtype>
void SqueezeCMomentCalc(const int n, const Dtype* wb, const Dtype* mask, Dtype* mu, Dtype* std, unsigned int* ncount){
  const unsigned int NUM_THREADS = 512;
  Dtype* pmu_g; Dtype* pstd_g; unsigned int* pncount_g;
  Dtype* pmu_c; Dtype* pstd_c; unsigned int* pncount_c;
  int num_p = (n+(NUM_THREADS<<1)-1)/(NUM_THREADS<<1);
  hipMalloc(&pmu_g, sizeof(Dtype)  * num_p);
  hipMalloc(&pstd_g, sizeof(Dtype) * num_p);
  hipMalloc(&pncount_g, sizeof(unsigned int) * num_p);
  pmu_c = (Dtype*) malloc(num_p * sizeof(Dtype));
  pstd_c = (Dtype*) malloc(num_p * sizeof(Dtype));
  pncount_c = (unsigned int*) malloc(num_p * sizeof(unsigned int));
  SqueezeCMomentCollect<Dtype><<<num_p,NUM_THREADS>>>(n, wb, mask, pmu_g, pstd_g, pncount_g);
  CUDA_POST_KERNEL_CHECK;
  hipMemcpy(pmu_c, pmu_g, sizeof(Dtype) * num_p, hipMemcpyDeviceToHost);
  hipMemcpy(pstd_c, pstd_g, sizeof(Dtype) * num_p, hipMemcpyDeviceToHost);
  hipMemcpy(pncount_c, pncount_g, sizeof(unsigned int) * num_p, hipMemcpyDeviceToHost);
  for (int i = 0; i < num_p; i++) {
    *mu += pmu_c[i];*std += pstd_c[i];*ncount += pncount_c[i];
  }
  hipFree(pmu_g);hipFree(pstd_g);hipFree(pncount_g);
  free(pmu_c);free(pstd_c);free(pncount_c);
}

template <typename Dtype>
void SqueezeInnerProductLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* weight = NULL;
  Dtype* weightMask = NULL;
  Dtype* weightTmp = NULL;
  const Dtype* bias = NULL;
  Dtype* biasMask = NULL;
  Dtype* biasTmp = NULL;
  Dtype* prune_threshold_params_gpu = NULL; // To store mu and std values
  Dtype* prune_threshold_params_cpu = NULL;
  prune_threshold_params_cpu = (Dtype*)malloc(sizeof(Dtype) * 2);
  int maskcount = 0;
  if (this->bias_term_) {
    weight = this->blobs_[0]->mutable_gpu_data();
    weightMask = this->blobs_[2]->mutable_gpu_data();
    weightTmp = this->weight_tmp_.mutable_gpu_data();
    bias = this->blobs_[1]->mutable_gpu_data();
    biasMask = this->blobs_[3]->mutable_gpu_data();
    prune_threshold_params_gpu = this->blobs_[4]->mutable_gpu_data();
    biasTmp = this->bias_tmp_.mutable_gpu_data();
    maskcount = this->blobs_[2]->count();
  }
  else {
    weight = this->blobs_[0]->mutable_gpu_data();
    weightMask = this->blobs_[1]->mutable_gpu_data();
    prune_threshold_params_gpu = this->blobs_[2]->mutable_gpu_data();
    weightTmp = this->weight_tmp_.mutable_gpu_data();
    maskcount = this->blobs_[1]->count();
  }

  if (this->phase_ == TRAIN){
    // Validate mask value to avoid corrupted mask value
    ValidateMask<Dtype><<<CAFFE_GET_BLOCKS(maskcount),
    CAFFE_CUDA_NUM_THREADS>>>( maskcount, weightMask);
    CUDA_POST_KERNEL_CHECK;

    // Calculate the mean and standard deviation of learnable parameters
    if (this->std == 0 && this->iter_ == 0){
      unsigned int ncount = 0;
      SqueezeCMomentCalc(this->blobs_[0]->count(), weight, weightMask, &this->mu, &this->std, &ncount);
      if (this->bias_term_) {
        SqueezeCMomentCalc(this->blobs_[1]->count(), bias, biasMask, &this->mu, &this->std, &ncount);
      }
      this->mu /= ncount; this->std -= ncount * this->mu * this->mu;
      this->std /= ncount; this->std = sqrt(this->std);
      prune_threshold_params_cpu[0] = this->mu;
      prune_threshold_params_cpu[1] = this->std;
      LOG(INFO)<<mu<<"  "<<std<<"  "<<ncount<<"\n";
      // Copy mu and std value from host to device
      hipMemcpy(prune_threshold_params_gpu, prune_threshold_params_cpu, sizeof(Dtype)*2, hipMemcpyHostToDevice);
    }
    // Copy mu and std value from Device to host
    hipMemcpy(prune_threshold_params_cpu, prune_threshold_params_gpu, sizeof(Dtype)*2, hipMemcpyDeviceToHost);
    // No pruning done during Retraining
    // Calculate the weight mask and bias mask with probability
    Dtype r = static_cast<Dtype>(rand())/static_cast<Dtype>(RAND_MAX);
    if (pow(1+(this->gamma)*(this->iter_),-(this->power))>r && (this->iter_)<(this->iter_stop_)) {
      SqueezeCMaskCalc<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
        CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[0]->count(), weight, weightMask, prune_threshold_params_cpu[0], prune_threshold_params_cpu[1], this->crate);
      CUDA_POST_KERNEL_CHECK;
      if (this->bias_term_) {
        SqueezeCMaskCalc<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[1]->count()),
          CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[1]->count(), bias, biasMask, prune_threshold_params_cpu[0], prune_threshold_params_cpu[1], this->crate);
        CUDA_POST_KERNEL_CHECK;
      }
    }
    // Dynamic Splicing
    // Unprune the pruned weights based on the splicing ratio
    if(this->dynamicsplicing)
    {
      if (this->iter_ == 0) {
        Dtype* weight_cpu = (Dtype *)malloc(this->blobs_[0]->count() *(sizeof(Dtype)));
        Dtype* weightMask_cpu = (Dtype *)malloc(this->blobs_[0]->count() *(sizeof(Dtype)));
        // Initially copy weight, weightMask to weight_cpu, weightMask_cpu and do Dynamic Splicing
        hipMemcpy(weight_cpu, weight, this->blobs_[0]->count() *(sizeof(Dtype)), hipMemcpyDeviceToHost);
        hipMemcpy(weightMask_cpu, weightMask, this->blobs_[0]->count() *(sizeof(Dtype)), hipMemcpyDeviceToHost);
        // Vector Pair holds weights and corresponding index for pruned nodes
        std::vector<std::pair<float, int> > prune_node;
        for (unsigned int k = 0; k < this->blobs_[0]->count(); ++k) {
          if(weightMask_cpu[k] == 0) {
            prune_node.push_back(make_pair(fabs(weight_cpu[k]), k));
          }
        }
        // Sort the weights and unprune the nodes
        std::sort(prune_node.begin(), prune_node.end());
        int zero_count = prune_node.size();
        int to_bespliced = zero_count * this->splicing_rate;
        int start_index = 0;
        int end_index = 0;
        for (unsigned int k = 0; k < zero_count; ++k) {
          if (prune_node[k].first > (0.25 * (prune_threshold_params_cpu[0] + prune_threshold_params_cpu[1]))) {
            start_index = k;
            break;
          }
        }
        if(start_index == 0)
          start_index = zero_count - to_bespliced; //Update start index
        end_index = start_index + to_bespliced;
        if (end_index > zero_count) {
          start_index = start_index - (end_index - zero_count);
          end_index = start_index + to_bespliced;
        }
        for (unsigned int k = start_index; k < end_index; ++k) {
          weightMask_cpu[prune_node[k].second] = 1;
        }
        hipMemcpy(weightMask, weightMask_cpu, this->blobs_[0]->count() *(sizeof(Dtype)), hipMemcpyHostToDevice);
        free(weightMask_cpu);
        free(weight_cpu);
        this->dynamicsplicing = false;
      }
    }
    free(prune_threshold_params_cpu);
  }

  // Calculate the current (masked) weight and bias
  SqueezeCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
    CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[0]->count(), weight, weightMask, weightTmp);
  CUDA_POST_KERNEL_CHECK;
  if (this->bias_term_) {
    SqueezeCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[1]->count()),
      CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[1]->count(), bias, biasMask, biasTmp);
    CUDA_POST_KERNEL_CHECK;
  }

  // Forward calculation with (masked) weight and bias
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  if (M_ == 1) {
    caffe_gpu_gemv<Dtype>(CblasNoTrans, N_, K_, (Dtype)1.,
                        weightTmp, bottom_data, (Dtype)0., top_data);
    if (this->bias_term_)
      caffe_gpu_axpy<Dtype>(N_, bias_multiplier_.cpu_data()[0],
                            biasTmp, top_data);
  } else {
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, M_, N_, K_, (Dtype)1.,
                          bottom_data, weightTmp, (Dtype)0., top_data);
    if (this->bias_term_)
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, (Dtype)1.,
                            bias_multiplier_.gpu_data(),
                            biasTmp, (Dtype)1., top_data);
  }
}

template <typename Dtype>
void SqueezeInnerProductLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  if (this->param_propagate_down_[0]) {
    const Dtype* weightMask = NULL;
    if(this->bias_term_)
      weightMask = this->blobs_[2]->gpu_data();
    else
      weightMask = this->blobs_[1]->gpu_data();
    Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    // Gradient with respect to weight
    SqueezeCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
      CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[0]->count(), weight_diff, weightMask, weight_diff);
    CUDA_POST_KERNEL_CHECK;
    caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, N_, K_, M_, (Dtype)1.,
        top_diff, bottom_data, (Dtype)1., weight_diff);
  }
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    const Dtype* biasMask = this->blobs_[3]->gpu_data();
    Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
    // Gradient with respect to bias
    SqueezeCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[3]->count()),
      CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[3]->count(), bias_diff, biasMask, bias_diff);
    CUDA_POST_KERNEL_CHECK;
    caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, (Dtype)1., top_diff,
        bias_multiplier_.gpu_data(), (Dtype)1.,bias_diff);
  }
  if (propagate_down[0]) {
    const Dtype* weightTmp = this->weight_tmp_.gpu_data();
    // Gradient with respect to bottom data
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, K_, N_, (Dtype)1.,
        top_diff, weightTmp, (Dtype)0.,
        bottom[0]->mutable_gpu_diff());
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SqueezeInnerProductLayer);

}  // namespace caffe
/***********************************************************************************************************************/
